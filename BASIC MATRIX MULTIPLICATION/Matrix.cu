#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
                   int numARows, int numAColumns,
                   int numBRows, int numBColumns,
                   int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
	int row = threadIdx.x + (blockDim.x*blockIdx.x);
	int col = threadIdx.y + (blockDim.y*blockIdx.y);
	
	if( row<numCRows && col<numCColumns )
	{
		int i;
		float sum = 0.0;
		for(i=0;i<numAColumns;i++)
		{
			sum = sum + (A[numAColumns*row + i]*B[numBColumns*i+col]);
		}
		C[numCColumns*row+col] = sum;
	}
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
	hostC = (float*)malloc(sizeof(float)*numCRows*numCColumns);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	wbCheck(hipMalloc((void**)&deviceA,sizeof(float)*numARows*numAColumns));
			
	wbCheck(hipMalloc((void**)&deviceB,sizeof(float)*numBRows*numBColumns));
			
	wbCheck(hipMalloc((void**)&deviceC,sizeof(float)*numCRows*numCColumns));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	
	hipMemcpy(deviceA,hostA,sizeof(float)*numARows*numAColumns,hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,sizeof(float)*numBRows*numBColumns,hipMemcpyHostToDevice);
	

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 block(16,16,1);
	dim3 grid(ceil(numARows/16.0),ceil(numBColumns/16.0),1);
	
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
	
	matrixMultiply<<<grid,block>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns);
	
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here

	hipMemcpy(hostC,deviceC,sizeof(float)*numCRows*numCColumns,hipMemcpyDeviceToHost);
	
	
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}


